#include "./common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>


void cudaInit(void){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
}
