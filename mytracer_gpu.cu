#include "./common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>

#include "Raytracer.h"
#include "utils/vec3.h"
#include "utils/Material.h"
#include "utils/Object.h"

void Raytracer::cudaInit(void){
    // Init Device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
}
